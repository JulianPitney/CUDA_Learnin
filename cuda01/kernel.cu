#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <thread>
#include <iostream>

using namespace std;


hipError_t addWithCuda(int *inputVec1, int *inputVec2, int *outputVec, unsigned int arraySize, dim3 gridDims, dim3 blockDims);

__global__ void addKernel(int *inputVec1, int *inputVec2, int *outputVec)
{
	
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    outputVec[i] = inputVec1[i] + inputVec2[i];
}

int main()
{
	unsigned int arraySize;
	cin >> arraySize;

	dim3 gridDims(65000);
	dim3 blockDims(1024);


	int *inputVec1 = new int[arraySize];
	int *inputVec2 = new int[arraySize];
	int *outputVec = new int[arraySize];

	auto t1 = std::chrono::high_resolution_clock::now();
	for (unsigned int i = 0; i < arraySize; i++)
	{
		inputVec1[i] = rand() % 354876;
		inputVec2[i] = rand() % 234587;
	}
	auto t2 = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
	cout << "Host Vec Init(ms): " << duration / 1000 << endl;



    hipError_t cudaStatus = addWithCuda(inputVec1, inputVec2, outputVec, arraySize, gridDims, blockDims);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }


    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t addWithCuda(int *inputVec1, int *inputVec2, int *outputVec, unsigned int arraySize, dim3 gridDims, dim3 blockDims)
{
    int *dev_vec1 = 0;
    int *dev_vec2 = 0;
    int *dev_outputVec = 0;
    hipError_t cudaStatus;

	auto t1_deviceSet = std::chrono::high_resolution_clock::now();
    cudaStatus = hipSetDevice(0);
	auto t2_deviceSet = std::chrono::high_resolution_clock::now();
	auto duration_deviceSet = std::chrono::duration_cast<std::chrono::microseconds>(t2_deviceSet - t1_deviceSet).count();
	cout << "Device Set Duration(ms): " << duration_deviceSet / 1000 << endl;;


    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	auto t1_malloc1 = std::chrono::high_resolution_clock::now();
    cudaStatus = hipMalloc((void**)&dev_vec1, arraySize * sizeof(int));
	auto t2_malloc1 = std::chrono::high_resolution_clock::now();
	auto duration_malloc1 = std::chrono::duration_cast<std::chrono::microseconds>(t2_malloc1 - t1_malloc1).count();
	cout << "Malloc1 Duration(ms): " << duration_malloc1 / 1000 << endl;;
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	auto t1_malloc2 = std::chrono::high_resolution_clock::now();
    cudaStatus = hipMalloc((void**)&dev_vec2, arraySize * sizeof(int));
	auto t2_malloc2 = std::chrono::high_resolution_clock::now();
	auto duration_malloc2 = std::chrono::duration_cast<std::chrono::microseconds>(t2_malloc2 - t1_malloc2).count();
	cout << "Malloc2 Duration(ms): " << duration_malloc2 / 1000 << endl;;
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	auto t1_malloc3 = std::chrono::high_resolution_clock::now();
    cudaStatus = hipMalloc((void**)&dev_outputVec, arraySize * sizeof(int));
	auto t2_malloc3 = std::chrono::high_resolution_clock::now();
	auto duration_malloc3 = std::chrono::duration_cast<std::chrono::microseconds>(t2_malloc3 - t1_malloc3).count();
	cout << "Malloc3 Duration(ms): " << duration_malloc3 / 1000 << endl;;
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	auto t1_memCpy1 = std::chrono::high_resolution_clock::now();
    cudaStatus = hipMemcpy(dev_vec1, inputVec1, arraySize * sizeof(int), hipMemcpyHostToDevice);
	auto t2_memCpy1 = std::chrono::high_resolution_clock::now();
	auto duration_memCpy1 = std::chrono::duration_cast<std::chrono::microseconds>(t2_memCpy1 - t1_memCpy1).count();
	cout << "MemCpy1 Duration(ms): " << duration_memCpy1 / 1000 << endl;;
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	auto t1_memCpy2 = std::chrono::high_resolution_clock::now();
    cudaStatus = hipMemcpy(dev_vec2, inputVec2, arraySize * sizeof(int), hipMemcpyHostToDevice);
	auto t2_memCpy2 = std::chrono::high_resolution_clock::now();
	auto duration_memCpy2 = std::chrono::duration_cast<std::chrono::microseconds>(t2_memCpy2 - t1_memCpy2).count();
	cout << "MemCpy2 Duration(ms): " << duration_memCpy2 / 1000 << endl;;
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	auto t1_kernelLaunch = std::chrono::high_resolution_clock::now();
    addKernel<<<gridDims, blockDims>>>(dev_vec1, dev_vec2, dev_outputVec);
	auto t2_kernelLaunch = std::chrono::high_resolution_clock::now();
	auto duration_kernelLaunch = std::chrono::duration_cast<std::chrono::microseconds>(t2_kernelLaunch - t1_kernelLaunch).count();
	cout << "Kernel Launch Duration(ms): " << duration_kernelLaunch / 1000 << endl;;

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	auto t3_kernelComplete = std::chrono::high_resolution_clock::now();
	auto duration_kernelRuntime = std::chrono::duration_cast<std::chrono::microseconds>(t3_kernelComplete - t1_kernelLaunch).count();
	cout << "Kernel Runtime(ms): " << duration_kernelRuntime / 1000 << endl;;



	auto t1_devToHost_memCpy = std::chrono::high_resolution_clock::now();
    cudaStatus = hipMemcpy(outputVec, dev_outputVec, arraySize * sizeof(int), hipMemcpyDeviceToHost);
	auto t2_devToHost_memCpy = std::chrono::high_resolution_clock::now();
	auto duration_devToHost_memCpy = std::chrono::duration_cast<std::chrono::microseconds>(t2_devToHost_memCpy - t1_devToHost_memCpy).count();
	cout << "devToHost memCpy Duration(ms): " << duration_devToHost_memCpy / 1000 << endl;;
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_outputVec);
    hipFree(dev_vec1);
    hipFree(dev_vec2);
    
    return cudaStatus;
}
